#include "hip/hip_runtime.h"
/*
 * ImageMultiplicationFilter.cpp
 *
 *  Created on: Aug 12, 2014
 *      Author: burnicki
 */

#include "ImageMultiplicationFilter.h"
#include "macros.h"

__global__ void ImageMultiplicationKernel(
		float *base_img,
		float *add_img,
		float *result,
		int image_width, int image_height) {
	// some neat index calculations:
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (IS_IN_IMAGE(x, y, image_width, image_height)) {
		int pixel_index = PIXEL_INDEX_OF(x, y, image_width);
		result[pixel_index] = base_img[pixel_index] * add_img[pixel_index] / 255.0;
	}
}

ImageMultiplicationFilter::ImageMultiplicationFilter(float *gpu_base_img) {
	gpu_base_img_ = gpu_base_img;
}

ImageMultiplicationFilter::~ImageMultiplicationFilter() {
	// TODO Auto-generated destructor stub
}

void ImageMultiplicationFilter::Run() {
	int imagew = GetImageWidth();
	int imageh = GetImageHeight();
	dim3 thread_block_size(32, 32, 1);
	dim3 block_grid_size(1 + imagew / thread_block_size.x,
			1 + imageh / thread_block_size.y,
			1);
	ImageMultiplicationKernel<<<block_grid_size, thread_block_size>>>(
			gpu_base_img_,
			GetGpuImageData(),
			GetGpuResultData(),
			imagew, imageh);
}
