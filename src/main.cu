#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cmath>

#include "ImagePencilFilter.h"
#include "JpegImage.h"
#include "ExpandableTexture.h"
#include "GrayscaleHistogram.h"

#include "ScetchFilter.h"
#include "ToneMappingFilter.h"
#include "ImageMultiplicationFilter.h"
#include "LogarithmicFilter.h"
#include "PotentialFilter.h"
#include "EquationSolver.h"


__global__ void convertRGBToYUV(float *outputImage, unsigned char* image, int image_size)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		int idx = pixel*RGB_COMPONENTS;
		float r = image[idx];
		float g = image[idx+1];
		float b = image[idx+2];
		float y = RGB_TO_Y(r, g, b);
		outputImage[idx] = y;
		outputImage[idx+1] = (b - y) * 0.493;
		outputImage[idx+2] = (r - y) * 0.877;

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

__global__ void GrayscaleAndYUVToRGB(unsigned char* outputImage, float *grayscaleImage, float *yuvImage, bool useColors, int image_size)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		int idx = pixel*RGB_COMPONENTS;
		float y = (float) grayscaleImage[pixel];
		float u = useColors ? (float) yuvImage[idx+1] : 0.0f;
		float v = useColors ? (float) yuvImage[idx+2] : 0.0f;

		float r = y+v/0.877;
		float b = y+u/0.493;
		float g = 1.703 * y - 0.509 * r - 0.194*b;

		// make sure that our values fit in a byte
		r = r < 0 ? 0 : r;
		g = g < 0 ? 0 : g;
		b = b < 0 ? 0 : b;

		r = r > 255 ? 255 : r;
		g = g > 255 ? 255 : g;
		b = b > 255 ? 255 : b;

		outputImage[idx] = r;
		outputImage[idx+1] = g;
		outputImage[idx+2] = b;

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

__global__ void extractGrayscale(float* grayscale, float *image, int image_size)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		grayscale[pixel] = image[pixel*RGB_COMPONENTS];

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

/**
 * \brief Kernel to calculate the forward gradient from a grayscale image
 *
 *        The bottom line and the very right line will be zero, as it is
 *        impossible to calculate the forward gradient for these points.
 *
 * \param kGrayscaleImage The input grayscale image
 * \param kImageSize      The size of the image in pixel
 * \param kImageWidth     The size of one line in the input image
 * \param gradient_image  Gradient output image
 *
 * TODO: Fix bank conflicts and do general optimization
 */
__global__ void CalculateGradientImage(
    const float *kGrayscaleImage,
    const int kImageSize,
    const int kImageWidth,
    float *gradient_image) {
  // Calculate pixel position
  int pixel_pos_this = blockDim.x * blockIdx.x + threadIdx.x;

  // Calculate gradient if pixel exists
  while (pixel_pos_this < kImageSize) {
    // Calculate forward pixels positions
    int pixel_pos_right = pixel_pos_this + 1;
    int pixel_pos_top   = pixel_pos_this + kImageWidth;

    // Set bottom and very right pixels to zero
    gradient_image[pixel_pos_this] = 0;

    // Calculate gradient if forward pixels exist
    if (pixel_pos_right < kImageSize && pixel_pos_top < kImageSize) {
      // Retrieve points value
      int pixel_this  = kGrayscaleImage[pixel_pos_this];
      int pixel_right = kGrayscaleImage[pixel_pos_right];
      int pixel_top   = kGrayscaleImage[pixel_pos_top];

      // Calculate difference between this and forward points
      int dx = pixel_right - pixel_this;
      int dy = pixel_top   - pixel_this;

      // Calculate the gradient for this point
      gradient_image[pixel_pos_this] =  sqrt(
          static_cast<float>( (dx * dx + dy * dy) )
      );
    }

    // Calculate next pixel position
    pixel_pos_this += MAX_BLOCKS * MAX_THREADS;
  }
}

void ExecutePipeline(const char *infilename, const char *outfilename, IPFConfiguration &config)
{
	/*
	 * CPU Preprocssing: Load image and texture from JPEG, set variables
	 */
	JpegImage cpuImage(infilename);
	ExpandableTexture pencilTexture(PENCIL_TEXTURE_PATH);

	int imageSize = cpuImage.PixelSize();
	int imageWidth = cpuImage.Width();
	int imageHeight = cpuImage.Height();

	/*
	 * GPU Setup: allocate buffers, set variables, upload image to GPU
	 */
	unsigned char * gpuCharImage;
	float * gpuFloatImage;
    float * gpuGrayscale;
	hipMalloc((void**) &gpuCharImage, cpuImage.ByteSize());
	hipMalloc((void**) &gpuFloatImage, imageSize * YUV_COMPONENTS * sizeof(float));
	hipMalloc((void**) &gpuGrayscale, imageSize * sizeof(float));
    dim3 blockGrid(MAX_BLOCKS);
    dim3 threadBlock(MAX_THREADS);

    hipMemcpy(gpuCharImage, cpuImage.Buffer(), cpuImage.ByteSize(), hipMemcpyHostToDevice);

    /*
     * GPU Preprocessing: Convert to YUV and extract Grayscale
     */
    std::cout << "Converting RGB to YUV" << std::endl;
    convertRGBToYUV<<<blockGrid, threadBlock>>>(gpuFloatImage, gpuCharImage, imageSize);

    std::cout << "Extracting grayscale Image" << std::endl;
    extractGrayscale<<<blockGrid, threadBlock>>>(gpuGrayscale, gpuFloatImage, imageSize);



    /*
     * Image 1: Create the scetched gradient image from Grayscale
     */
    float *gpu_gradient_image;
    hipMalloc((void**) &gpu_gradient_image, imageSize * sizeof(float));

    std::cout << "Calculating the Gradient" << std::endl;
    CalculateGradientImage<<<blockGrid, threadBlock>>>(
        gpuGrayscale,
        imageSize,
        imageWidth,
        gpu_gradient_image);

    std::cout << "Calculating the scetch filter" << std::endl;
    ScetchFilter scetch_filter(config);
    scetch_filter.SetImageFromGpu(gpu_gradient_image, imageWidth, imageHeight);
    scetch_filter.Run();


    /*
     * Image 2: Create the textured tone-mapped image from Grayscale
     */
    std::cout << "Calculating the target tone map on CPU" << std::endl;
    ToneMap targetToneMap(config);

    std::cout << "Calculating the histogram of the grayscale image" << std::endl;
    GrayscaleHistogram histogram(gpuGrayscale, imageSize);
    histogram.Run();

    std::cout << "Calculating the tone mapping filter" << std::endl;
    ToneMappingFilter tone_filter(targetToneMap, histogram.GpuCummulativeHistogram());
    tone_filter.SetImageFromGpu(gpuGrayscale, imageWidth, imageHeight);
    tone_filter.Run();

    std::cout << "Calculate the log of tonemapped image" << std::endl;
    LogarithmicFilter log_filter;
    log_filter.SetImageFromGpu(tone_filter.GetGpuResultData(), imageWidth, imageHeight);
    log_filter.Run();

    std::cout << "Expanding and apply log function to texture on CPU" << std::endl;
    pencilTexture.Expand(imageWidth, imageHeight);

    std::cout << "Solving equation for texture drawing" << std::endl;
    EquationSolver equation_solver(pencilTexture.LogBuffer(), log_filter.GetCpuResultData(),
    		imageWidth, imageHeight, config.TextureRenderingSmoothness);
    equation_solver.Run();
    float *beta_star = equation_solver.GetResult();

    std::cout << "Rendering computed texture" << std::endl;
    PotentialFilter potential_filter(beta_star);
    potential_filter.SetImageFromCpu(pencilTexture.ExpandedBuffer(), imageWidth, imageHeight);
    potential_filter.Run();



    /*
     * Combined Image: Multiplying texture tone-mapped image with scetched gradient image
     */
    std::cout << "Multiplicating both images" << std::endl;
    ImageMultiplicationFilter image_multiplication(scetch_filter.GetGpuResultData());
    image_multiplication.SetImageFromGpu(potential_filter.GetGpuResultData(), imageWidth, imageHeight);
    image_multiplication.Run();

    float *resultGrayscaleImage = image_multiplication.GetGpuResultData();


    /*
     * GPU Postprocessing: Convert to RGB, either with colors or without
     */
    GrayscaleAndYUVToRGB<<<blockGrid, threadBlock>>>(gpuCharImage, resultGrayscaleImage, gpuFloatImage, config.UseColors, imageSize);


    /*
     * CPU Postprocessing: Download image and save it as JPEG
     */


    hipMemcpy(cpuImage.Buffer(), gpuCharImage, cpuImage.ByteSize(), hipMemcpyDeviceToHost);
    cpuImage.Save(outfilename);
	std::cout << "Done." << std::endl;

	/*
	 * Cleanup
	 */
	hipFree(gpu_gradient_image);
	hipFree(gpuGrayscale);
	hipFree(gpuFloatImage);
	hipFree(gpuCharImage);
}





int main(int argc, char* argv[]) {
	if (argc < 3)
	{
		std::cout << "Please provide input and output filenames as arguments." << std::endl;
		return 1;
	}
	IPFConfiguration config;
	config.UseColors = !(argc > 3 && strcmp(argv[3], "-grayscale") == 0);

	try
	{
		ExecutePipeline(argv[1], argv[2], config);
	}
	catch (const char *msg)
	{
		std::cout << msg << std::endl;
		return 1;
	}

	return 0;
}
