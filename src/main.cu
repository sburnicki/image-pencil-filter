#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>

#include "../lib/jpge.h"
#include "../lib/jpgd.h"

#include "ScetchFilter.h"

#define MAX_BLOCKS 256
#define MAX_THREADS 256

__global__ void convertRGBToYUV(float *outputImage, unsigned char* image, int image_size, int comps)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		int idx = pixel*comps;
		float r = (float) image[idx];
		float g = (float) image[idx+1];
		float b = (float) image[idx+2];
		float y = 0.299 * r + 0.587 * g + 0.114 * b;
		outputImage[idx] = y;
		outputImage[idx+1] = (b - y) * 0.493;
		outputImage[idx+2] = (r - y) * 0.877;

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

__global__ void convertYUVToRGB(unsigned char* outputImage, float *image, int image_size, int comps)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		int idx = pixel*comps;
		float y = (float) image[idx];
		float u = (float) image[idx+1];
		float v = (float) image[idx+2];
		float r = y+v/0.877;
		float b = y+u/0.493;
		outputImage[idx] = r;
		outputImage[idx+1] = 1.704 * y - 0.509 * r - 0.194*b;
		outputImage[idx+2] = b;

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

/**
 * \brief Kernel to transform the gradient image into a RGB image
 *
 *        This kernel is only for testing the gradient image output.
 *
 * \param kGradientImage  The input gradient image
 * \param kImageSize      The size of the image in pixel
 * \param rgb_image       RGB output image
 */
__global__ void ConvertGradienToRGB(
    const float *kGradientImage,
    const int kImageSize,
    const int kImageComponents,
    unsigned char *rgb_image) {
  // Calculate pixel position
  int pixel_pos_this = blockDim.x * blockIdx.x + threadIdx.x;

  // Calculate RGB value if pixel exists
  while (pixel_pos_this < kImageSize)
  {
    // Transform to YUV
    float y = kGradientImage[pixel_pos_this];
    float u = 0.0;
    float v = 0.0;

    // Calculate RGB
    float r = y + v / 0.877;
    float b = y + u / 0.493;

    // Save RGB
    int pixel_pos_output = pixel_pos_this * kImageComponents;
    rgb_image[pixel_pos_output]     = r;
    rgb_image[pixel_pos_output + 1] = 1.704 * y - 0.509 * r - 0.194 * b;
    rgb_image[pixel_pos_output + 2] = b;

    // Calculate next pixel position
    pixel_pos_this += MAX_BLOCKS * MAX_THREADS;
  }
}

__global__ void extractGrayscale(float* grayscale, float *image, int image_size, int comps)
{
	int pixel = blockDim.x * blockIdx.x + threadIdx.x;
	while (pixel < image_size)
	{
		grayscale[pixel] = image[pixel*comps];

		pixel += MAX_BLOCKS * MAX_THREADS;
	}
}

/**
 * \brief Kernel to calculate the forward gradient from a grayscale image
 *
 *        The bottom line and the very right line will be zero, as it is
 *        impossible to calculate the forward gradient for these points.
 *
 * \param kGrayscaleImage The input grayscale image
 * \param kImageSize      The size of the image in pixel
 * \param kImageWidth     The size of one line in the input image
 * \param gradient_image  Gradient output image
 *
 * TODO: Fix bank conflicts and do general optimization
 */
__global__ void CalculateGradientImage(
    const float *kGrayscaleImage,
    const int kImageSize,
    const int kImageWidth,
    float *gradient_image) {
  // Calculate pixel position
  int pixel_pos_this = blockDim.x * blockIdx.x + threadIdx.x;

  // Calculate gradient if pixel exists
  while (pixel_pos_this < kImageSize) {
    // Calculate forward pixels positions
    int pixel_pos_right = pixel_pos_this + 1;
    int pixel_pos_top   = pixel_pos_this + kImageWidth;

    // Set bottom and very right pixels to zero
    gradient_image[pixel_pos_this] = 0;

    // Calculate gradient if forward pixels exist
    if (pixel_pos_right < kImageSize && pixel_pos_top < kImageSize) {
      // Retrieve points value
      int pixel_this  = kGrayscaleImage[pixel_pos_this];
      int pixel_right = kGrayscaleImage[pixel_pos_right];
      int pixel_top   = kGrayscaleImage[pixel_pos_top];

      // Calculate difference between this and forward points
      int dx = pixel_right - pixel_this;
      int dy = pixel_top   - pixel_this;

      // Calculate the gradient for this point
      gradient_image[pixel_pos_this] =  sqrt(
          static_cast<float>( (dx * dx + dy * dy) )
      );
    }

    // Calculate next pixel position
    pixel_pos_this += MAX_BLOCKS * MAX_THREADS;
  }
}

/**
 * \brief Kernel to calculate the histogram from a grayscale image
 * \param kGrayscaleImage         The input grayscale image
 * \param kImageSize              The size of the image in pixel
 * \param histogram               Histogram output
 * \param accumulative_histogramm Accumulative histogram output
 *
 * TODO: Check this out - http://developer.download.nvidia.com/compute/cuda/\
 *                        1.1-Beta/x86_website/projects/histogram64/doc/\
 *                        histogram.pdf
 */
__global__ void CalculateHistogram(
    const float *kGrayscaleImage,
    const int kImageSize,
    int *histogram,
    int *accumulative_histogram) {
  __shared__ int shared_histogram[256];
  __shared__ int shared_accumulative_histogram[256];

  // Calculate ID and pixel position
  int tid       = threadIdx.x;
  int pixel_pos = blockDim.x * blockIdx.x + tid;

  // Clear histogram
  if (pixel_pos < 256) {
      histogram[pixel_pos] = 0;
      accumulative_histogram[pixel_pos] = 0;
  }
  if (tid < 256) {
    shared_histogram[tid] = 0;
    shared_accumulative_histogram[tid] = 0;
  }
  __syncthreads();

  // Calculate partial histogram if pixel exists
  while (pixel_pos < kImageSize) {
    int value = kGrayscaleImage[pixel_pos];

    // Increment position of value in histogram
    // TODO Remove sanity check if sure
    if (value < 256 && value >= 0) {
      atomicAdd(&shared_histogram[value], 1);
    }

    // Calculate next pixel position
    pixel_pos += MAX_BLOCKS * MAX_THREADS;
  }
  __syncthreads();

  // Calculate partial histogram and accumulate result to global memory
  if (tid < 256) {
    shared_accumulative_histogram[tid] = shared_histogram[tid];

    // TODO: Fix the commented code block and delete the uncommented slower one
    //       The result is too big in this faster solution
    //for (int i = 1; i <= tid; i *= 2) {
    //  __syncthreads();
    //  shared_accumulative_histogram[tid]
    //      += shared_accumulative_histogram[tid - i];
    //}
    __syncthreads();
    int sum = 0;
    for (int i = 0; i <= tid; i++) {
      sum += shared_accumulative_histogram[i];
    }
    __syncthreads();
    shared_accumulative_histogram[tid] = sum;

    // Copy result to global memory
    __syncthreads();
    atomicAdd(&histogram[tid], shared_histogram[tid]);
    atomicAdd(&accumulative_histogram[tid], shared_accumulative_histogram[tid]);
  }
}

int main(int argc, char* argv[]) {
	int width, height, comps, image_size;

	if (argc < 3)
	{
		std::cout << "Please provide input and output filenames as arguments." << std::endl;
		return 1;
	}
	char *infilename = argv[1];
	char *outfilename = argv[2];

	// load image, allocate space on GPU
	unsigned char * image = jpgd::decompress_jpeg_image_from_file(infilename, &width, &height, &comps, 3);
	image_size = width * height;
	if (comps != 3)
	{
		if (comps == 0)
		{
			std::cout << "Loading the image failed! Wrong path?." << std::endl;
		}
		else
		{
			std::cout << "Currently only images with 3 components are supported." << std::endl;
		}
		free(image);
		return 1;
	}

	unsigned char * gpuCharImage;
	float * gpuFloatImage;
    float * gpuGrayscale;
	hipMalloc((void**) &gpuCharImage, image_size * comps * sizeof(unsigned char));
	hipMalloc((void**) &gpuFloatImage, image_size * comps * sizeof(float));
	hipMalloc((void**) &gpuGrayscale, image_size * sizeof(float));

	// upload to gpu
    hipMemcpy(gpuCharImage, image, image_size * comps * sizeof(unsigned char), hipMemcpyHostToDevice);

	// convert to YUV
    dim3 blockGrid(MAX_BLOCKS);
    dim3 threadBlock(MAX_THREADS);
    std::cout << "Converting RGB to YUV" << std::endl;
    convertRGBToYUV<<<blockGrid, threadBlock>>>(gpuFloatImage, gpuCharImage, image_size, comps);

    // extract grayscale
    std::cout << "Extracting grayscale Image" << std::endl;
    extractGrayscale<<<blockGrid, threadBlock>>>(gpuGrayscale, gpuFloatImage, image_size, comps);

    // Calculate gradient image
    float *gpu_gradient_image;
    hipMalloc((void**) &gpu_gradient_image, image_size * sizeof(float));

    std::cout << "Calculating the Gradient" << std::endl;
    CalculateGradientImage<<<blockGrid, threadBlock>>>(
        gpuGrayscale,
        image_size,
        width,
        gpu_gradient_image);



    std::cout << "Calculating the scetch filter" << std::endl;
    // Apply Scetch Filter
    ScetchFilter scetch_filter;
    scetch_filter.SetImageFromGpu(gpu_gradient_image, width, height);
    scetch_filter.set_line_count(7);
    scetch_filter.set_line_length(20);
    scetch_filter.set_line_strength(1);
    scetch_filter.set_gamma(1);
    scetch_filter.Run();


//    std::cout << "Running scetch filter debug tests" << std::endl;
//    std::string debug_message;
//    if (!scetch_filter.TestGpuFunctions(&debug_message)) {
//    	std::cerr << "scetch filter test failed with message:" << std::endl <<
//    			debug_message << std::endl;
//    }

    std::cout << "Create rgb image from greyscale image" << std::endl;

    // Calculate histogram
    int * gpu_histogram;
    int * gpu_accumulative_histogram;
    hipMalloc((void**) &gpu_histogram, 256 * sizeof(int));
    hipMalloc((void**) &gpu_accumulative_histogram, 256 * sizeof(int));

    std::cout << "Calculating the histogram of the grayscale image"
              << std::endl;
    CalculateHistogram<<<blockGrid, threadBlock>>>(
        gpuGrayscale,
        image_size,
        gpu_histogram,
        gpu_accumulative_histogram);

    hipDeviceSynchronize();

    // TODO: Only for testing purpose, remove for production
    int histogram[256];
    int accumulative_histogram[256];
    hipMemcpy(
        &histogram,
        gpu_histogram,
        256 * sizeof(int),
        hipMemcpyDeviceToHost);
    hipMemcpy(
        &accumulative_histogram,
        gpu_accumulative_histogram,
        256 * sizeof(int),
        hipMemcpyDeviceToHost);
    std::cout << "Histogram: ";
    for (int i = 0; i <= 255; i++) {
      std::cout << " " << histogram[i];
    }
    std::cout << std::endl;
    std::cout << "Accumulative histogram: ";
    for (int i = 0; i <= 255; i++) {
      std::cout << " " << accumulative_histogram[i];
    }
    std::cout << std::endl;
    int sum = 0;
    for (int i = 0; i <= 255; i++) {
      sum += histogram[i];
    }
    std::cout << "Control Sum: " << sum << std::endl;

    // Output grayscale image
    ConvertGradienToRGB<<<blockGrid, threadBlock>>>(
        scetch_filter.GetGpuResultData(),
        image_size,
        comps,
        gpuCharImage);


	// convert to RGB
    //convertYUVToRGB<<<blockGrid, threadBlock>>>(gpuCharImage, gpuFloatImage, image_size, comps);

	// download image
    hipMemcpy(image, gpuCharImage, image_size * comps * sizeof(unsigned char), hipMemcpyDeviceToHost);

	// write image
	if(!jpge::compress_image_to_jpeg_file(outfilename, width, height, comps, image))
	{
		std::cout << "Error writing the image." << std::endl;
	}

	free(image);
	hipFree(gpu_histogram);
	hipFree(gpu_accumulative_histogram);
	hipFree(gpu_gradient_image);
	hipFree(gpuGrayscale);
	hipFree(gpuFloatImage);
	hipFree(gpuCharImage);
}
