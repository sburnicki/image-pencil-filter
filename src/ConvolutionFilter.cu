#include "hip/hip_runtime.h"
/*
 * ConvolutionFilter.cpp
 *
 *  Created on: May 12, 2014
 *      Author: braunra
 */

#include "ConvolutionFilter.h"
#include <stdio.h>
#include <stdlib.h>

// Used Kernel functions

__global__ void SimpleConvolutionKernel(const float* source_image,
										const float* kernel,
										float* result,
										int image_width,
										int image_height,
										int kernel_width, int kernel_height) {
	// TODO

}


ConvolutionFilter::ConvolutionFilter() {
	image_width_ = image_height_ = 0;
	free_image_ = free_kernel_ = false;
}


ConvolutionFilter::~ConvolutionFilter() {
	hipFree(gpu_result_data_);
	if (free_image_)
		hipFree(gpu_image_data_);
	if (free_kernel_)
		hipFree(gpu_kernel_data_);
}

void ConvolutionFilter::SetImage(float* cpu_image_data, int image_width,
		int image_height) {
	image_width_ = image_width;
	image_height_ = image_height;
	// allocate gpu memory
	hipMalloc((void**) &gpu_image_data_, image_byte_count());
	hipMalloc((void**) &gpu_result_data_, image_byte_count());
	// copy data to gpu
	hipMemcpy(gpu_image_data_, cpu_image_data, image_byte_count(),
			hipMemcpyHostToDevice);
}

void ConvolutionFilter::UseImage(float* gpu_image_data, int image_width,
		int image_height) {
	image_width_ = image_width;
	image_height_ = image_height;
	gpu_image_data_ = gpu_image_data;
	hipMalloc((void**) &gpu_result_data_, image_byte_count());
}

void ConvolutionFilter::SetKernel(float* cpu_kernel_data,
		int kernel_width, int kernel_height) {
	kernel_width_ = kernel_width;
	kernel_height_ = kernel_height;
	// allocate gpu memory
	hipMalloc((void**) &gpu_kernel_data_, image_byte_count());
	// copy data to gpu
	hipMemcpy(gpu_kernel_data_, cpu_kernel_data, kernel_byte_count(),
			hipMemcpyHostToDevice);
}

void ConvolutionFilter::UseKernel(float* gpu_kernel_data,
		int kernel_width, int kernel_height) {
	kernel_width_ = kernel_width;
	kernel_height_ = kernel_height;
	gpu_kernel_data_ = gpu_kernel_data;
}

int ConvolutionFilter::image_pixel_count() {
	return image_height_ * image_width_;
}

int ConvolutionFilter::kernel_pixel_count() {
	return kernel_height_ * kernel_width_;
}

int ConvolutionFilter::image_byte_count() {
	return image_pixel_count() * sizeof(float);
}

void ConvolutionFilter::Run() {
	dim3 thread_block_size(256, 256, 1);
	dim3 block_grid_size(1 + image_width_ / thread_block_size.x,
						 1 + image_height_ / thread_block_size.y,
						 1);
	SimpleConvolutionKernel<<<block_grid_size, thread_block_size>>>(
			gpu_image_data_,
			gpu_kernel_data_,
			gpu_result_data_,
			image_width_, image_height_,
			kernel_width_, kernel_height_);
}

int ConvolutionFilter::kernel_byte_count() {
	return kernel_pixel_count() * sizeof(float);
}
