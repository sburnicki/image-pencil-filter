/*
 * ConvolutionFilter.cpp
 *
 *  Created on: May 12, 2014
 *      Author: braunra
 */

#include "ConvolutionFilter.h"
#include <stdio.h>
#include <stdlib.h>

ConvolutionFilter::ConvolutionFilter() {
	image_width_ = image_height_ = 0;
}


ConvolutionFilter::~ConvolutionFilter() {
	// TODO Auto-generated destructor stub
}

void ConvolutionFilter::SetImage(float* cpu_image_data, int image_width,
		int image_height) {
	image_width_ = image_width;
	image_height_ = image_height;
	// allocate gpu memory
	hipMalloc((void**) &gpu_image_data_, image_byte_count());
	hipMalloc((void**) &gpu_result_data_, image_byte_count());
	// copy data to gpu
	hipMemcpy(gpu_image_data_, cpu_image_data, image_byte_count(),
			hipMemcpyHostToDevice);
}

void ConvolutionFilter::UseImage(float* gpu_image_data, int image_width,
		int image_height) {
	image_width_ = image_width;
	image_height_ = image_height;
	gpu_image_data_ = gpu_image_data;
	hipMalloc((void**) &gpu_result_data_, image_byte_count());
}

void ConvolutionFilter::SetKernel(float* cpu_kernel_data,
		int kernel_width, int kernel_height) {
	kernel_width_ = kernel_width;
	kernel_height_ = kernel_height;
	// allocate gpu memory
	hipMalloc((void**) &gpu_kernel_data_, image_byte_count());
	// copy data to gpu
	hipMemcpy(gpu_kernel_data_, cpu_kernel_data, kernel_byte_count(),
			hipMemcpyHostToDevice);
}

void ConvolutionFilter::UseKernel(float* gpu_kernel_data,
		int kernel_width, int kernel_height) {
	kernel_width_ = kernel_width;
	kernel_height_ = kernel_height;
	gpu_kernel_data_ = gpu_kernel_data;
}

int ConvolutionFilter::image_pixel_count() {
	return image_height_ * image_width_;
}

int ConvolutionFilter::kernel_pixel_count() {
	return kernel_height_ * kernel_width_;
}

int ConvolutionFilter::image_byte_count() {
	return image_pixel_count() * sizeof(float);
}

int ConvolutionFilter::kernel_byte_count() {
	return kernel_pixel_count() * sizeof(float);
}
