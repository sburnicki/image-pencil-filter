#include "hip/hip_runtime.h"
/*
 * ConvolutionFilter.cpp
 *
 *  Created on: May 12, 2014
 *      Author: braunra
 */

#include "ScetchFilter.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <vector>

// only for debugging!
#include "../lib/jpge.h"
#include "../lib/jpgd.h"
// -------------------

// shared Memory Size per Multiprocessor is 48KB with Cuda 2.0 to 4.x
// 48KB equals 48k/4 floats
// the side lenght of a shared memory block is the squareroot of 40k/4
// sqrt(48000 / 4) = 109.544...
#define SHARED_2D_BLOCK_DIMENSION 109

// Used Kernel functions
__device__ __host__ int clamp(int value, int mi, int ma) {
  return max(mi, min(ma, value));
}

__device__ __host__ int PixelIndexOf(int x, int y, int width) {
  return x + y * width;
}

__device__ __host__ bool IsInImage(int x, int y, int width, int height) {
  return x >= 0 && x < width &&
    y >= 0 && y < height;
}

__device__ __host__ bool IsInSharedMemoryBlock(int x, int y, int block_dim) {
  return x >= 0 && y >= 0 &&
    x < block_dim && y < block_dim;
}

__device__ __host__ void RotatedCoordinate(float *x, float *y, float angle) {
  float c = cos(angle);
  float s = sin(angle);
  float new_x = c * (*x) - s* (*y);
  float new_y = s * (*x) + c * (*y);
  (*x) = new_x;
  (*y) = new_y;
}


// calculates indices and corresponding weights of all pixels along a line
__device__ __host__ int LinePixels(int x, int y, float line_angle, int image_width, int image_height,
    int line_length, float line_strength,
    int *indices, float *weights) {
  int line_pixel_count = 0;
  float halve_length = static_cast<float>(line_length) / 2.f;
  float halve_strength = line_strength / 2.f;

  for (int j = ceil(y - halve_strength); j < ceil(y + halve_strength); j++) {
    for (int i = ceil(x - halve_length); i < ceil(x + halve_length); i++) {
      float rotated_x = i - x;
      float rotated_y = j - y;
      RotatedCoordinate(&rotated_x, &rotated_y, line_angle);
      rotated_x  += x;
      rotated_y  += y;
      if (IsInImage(rotated_x, rotated_y, image_width, image_height)) {
        indices[line_pixel_count] = PixelIndexOf(rotated_x, rotated_y, image_width);
        weights[line_pixel_count] = 1;
        line_pixel_count++;
      }
    }
  }
  return line_pixel_count;
}

// scetch kernel
__global__ void SimpleScetchKernel(
    float *image,
    float *result,
    int image_width, int image_height,
    int line_length, float line_strength, int line_count,
    float gamma) {
  // some neat index calculations:
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;

  if (IsInImage(x, y, image_width, image_height)) {
    int pixel_index = PixelIndexOf(x, y, image_width);

    // the number of pixels in a line equals the number of pixels in a rectangle
    // the true number of pixels might be smaller due to image boundaries
    int max_line_pixel_count = line_strength * line_length;

    // allocate some memory for the line pixel indices and the corresponding weights
    int* line_pixel_indices = new int[max_line_pixel_count];
    float* weights = new float[max_line_pixel_count];
    float max_value = 0.f;
    for (int line = 0; line < line_count; line++) {
      float line_angle = static_cast<float>(line) * (M_PI / line_count); // Random?
      int line_pixel_count = LinePixels(x, y, line_angle, image_width, image_height,
          line_length, line_strength,
          line_pixel_indices, weights);
      float convolution_result = 0;
      for  (int i = 0; i < line_pixel_count; i++) {
        float line_pixel_value = image[line_pixel_indices[i]];
        convolution_result += line_pixel_value * weights[i] / line_pixel_count;
      }
      max_value = max(max_value, convolution_result);
    }

    delete[] line_pixel_indices;
    delete[] weights;
    result[pixel_index] = max(255.f - __powf(max_value, gamma), 0.f);
  }
}

__device__ __host__ bool CalculateCoordinatesInSharedMemoryBlock(
    int x, int y,
    int thread_x, int thread_y,
    int image_x, int image_y,
    float rotation_angle,
    int half_length,
    int shared_width,
    int image_width, int image_height,
    int *shared_x, int *shared_y) {
  float start_x = -half_length;
  float start_y = 0;
  float current_x = start_x + x;
  float current_y = start_y + y;
  RotatedCoordinate(&current_x, &current_y, rotation_angle);
  int rotated_image_x = current_x + image_x;
  int rotated_image_y = current_y + image_y;
  current_x += half_length;  // in shared memory (0,0) is located at
  current_y += half_length;  // (half_length, half_length)
  *shared_x = current_x + thread_x;
  *shared_y = current_y + thread_y;
  return IsInSharedMemoryBlock(*shared_x, *shared_y, shared_width) &&
         IsInImage(rotated_image_x, rotated_image_y, image_width, image_height);
}

__device__ __host__ void ImageCoordinatesFromSharedAddress(
    int shared_address,
    int shared_width,
    int start_x,
    int start_y,
    int *image_x,
    int *image_y) {
  *image_x = shared_address % shared_width;
  *image_y = shared_address / shared_width;
  *image_x = (*image_x) + start_x;
  *image_y = (*image_y) + start_y;
}



// fast scetch kernel
__global__ void HighSpeedScetchKernel(
    float *image,
    float *result,
    int image_width,
    int image_height,
    int shared_width,
    int line_length,
    float line_strength,
    int line_count,
    float rotation_offset,
    float gamma) {
  // Create a shared memory block
  extern __shared__ float image_block[];

  int overhang = ceil(static_cast<float>(line_length) / 2.f);  //TODO(Raphael) was wenn line_length ungerade?
  int x_image = threadIdx.x + blockDim.x * blockIdx.x;
  int y_image = threadIdx.y + blockDim.y * blockIdx.y;

  int thread_number = threadIdx.x + blockDim.x * threadIdx.y;
  int thread_count_in_block = blockDim.x * blockDim.y;
  int num_copy_iterations = ceil(static_cast<float>(shared_width * shared_width) /
      thread_count_in_block);
  int start_x = blockDim.x * blockIdx.x - overhang;
  int start_y = blockDim.y * blockIdx.y - overhang;
  for (int i = 0; i < num_copy_iterations; i++) {
    int shared_address = thread_number + i * thread_count_in_block;
    if (shared_address < shared_width * shared_width) {
      int x, y;
      ImageCoordinatesFromSharedAddress(
          shared_address,
          shared_width,
          start_x,
          start_y,
          &x,
          &y);
      if (IsInImage(x, y, image_width, image_height))
        image_block[shared_address] = image[PixelIndexOf(x, y, image_width)];
      else // TODO(Raphael) debug!
        image_block[shared_address] = 0.f;
    }
  }
  __syncthreads();

  if (IsInImage(x_image, y_image, image_width, image_height)) {
    // calculate line convolution for all directions
    float angle_step = M_PI / line_count;
    float max_convolution_result = 0.f;
    for (int line_index = 0; line_index < line_count; line_index++) {
      float rotation_angle = angle_step * line_index;
      int n_pixels = 0;
      float sum = 0.f;
      // move along the line from left to right and collect the pixel values
      for (int y = 0; y < line_strength; y++) {
        for (int x = 0; x < line_length; x++) {
          int shared_x, shared_y;
          bool is_inside_block = CalculateCoordinatesInSharedMemoryBlock(
              x, y,
              threadIdx.x, threadIdx.y,
              x_image, y_image,
              rotation_angle,
              overhang,
              shared_width,
              image_width, image_height,
              &shared_x, &shared_y);
          if (is_inside_block) {
            sum = sum + image_block[PixelIndexOf(shared_x, shared_y, shared_width)];
            n_pixels += 1;
          }
        }
      }
      // do the convolution and take the line if its the best so far
      max_convolution_result = max(max_convolution_result, sum / n_pixels);
    }
    // calculate gamma
    result[PixelIndexOf(x_image, y_image, image_width)] =
      max(255.f - __powf(max_convolution_result, gamma), 0.f);
  }
}


ScetchFilter::ScetchFilter() : ImageFilter() {
  line_length_ = 20;
  line_strength_ = 1;
  line_count_  = 4;
  gamma_ = 1.f;
  rotation_offset_ = 0.f;
}


void ScetchFilter::set_line_strength(float line_strength) {
  line_strength_ = line_strength;
}

void ScetchFilter::set_line_length(int line_length) {
  line_length_ = line_length;
}

void ScetchFilter::set_line_count(int line_count) {
  line_count_ = line_count;
}

void ScetchFilter::set_line_rotation_offset(float offset_angle) {
  rotation_offset_ = offset_angle;
}

void ScetchFilter::set_gamma(float gamma) {
  gamma_ = gamma;
}

void ScetchFilter::Run() {
/*
     int imageh = GetImageHeight();
     int imagew = GetImageWidth();
     dim3 thread_block_size(32, 32, 1);
     dim3 block_grid_size(1 + imagew / thread_block_size.x,
     1 + imageh / thread_block_size.y,
     1);
     SimpleScetchKernel<<<block_grid_size, thread_block_size>>>(
     GetGpuImageData(),
     GetGpuResultData(),
     imagew, imageh,
     line_length_, line_strength_, line_count_,
     gamma_);
*/
  // Max threads per Block = 1024 ==> sqrt(1024) = 32
  int pixels_per_dimension = min(SHARED_2D_BLOCK_DIMENSION - (line_length_ + 1), 32);
  dim3 high_speed_block_size(pixels_per_dimension, pixels_per_dimension, 1);
  dim3 high_speed_grid_size(GetImageWidth() / pixels_per_dimension + 1,
      GetImageHeight() / pixels_per_dimension + 1,
      1);
  int memory_per_dimension = pixels_per_dimension + line_length_ + 1;
  int shared_memory_size = sizeof(float) * memory_per_dimension * memory_per_dimension;
  HighSpeedScetchKernel<<<high_speed_grid_size, high_speed_block_size, shared_memory_size>>>(
      GetGpuImageData(),
      GetGpuResultData(),
      GetImageWidth(),
      GetImageHeight(),
      memory_per_dimension,
      line_length_,
      line_strength_,
      line_count_,
      rotation_offset_,
      gamma_);
}

/*

bool ScetchFilter::TestGpuFunctions(std::string *message,
    std::string *additional_message, bool *is_additional_message) {
  int** lines = new int*[line_count_];
  float** weights = new float*[line_count_];
  int max_line_pixel_count = line_strength_ * line_length_;
  int image_width = GetImageWidth();
  int image_height = GetImageHeight();

  // check if max_line_pixel_count is big enough
  for (int x = 0; x < image_width; x++) {
    for (int y = 0; x < image_width; x++) { // <<< wtf, is this correct?
      for (int i = 0; i < line_count_; i++) {
        lines[i] = new int[max_line_pixel_count];
        weights[i] = new float[max_line_pixel_count];
        float line_anle = static_cast<float>(i) * (M_PI / line_count_);
        int line_pixels_count = LinePixels(x, y, line_anle, image_width, image_height,
            line_length_, line_strength_,
            lines[i], weights[i]);
        if (line_pixels_count > max_line_pixel_count) {
          char x_string[16], y_string[16];
          sprintf(x_string, "%d", x);
          sprintf(y_string, "%d", y);
          (*message) = std::string("ERROR: more LinePixels returnt to many pixels for position (") +
            x_string + "," + y_string + ")!";
          return false;
        }
      }
    }
  }

  // show lines for some pixels
  for (int i = 0; i < line_count_; i++) {
    lines[i] = new int[max_line_pixel_count];
    weights[i] = new float[max_line_pixel_count];
    float line_anle = static_cast<float>(i) * (M_PI / line_count_);
    int line_pixels_count = LinePixels(100, 100, line_anle, image_width, image_height,
        line_length_, line_strength_,
        lines[i], weights[i]);

    // create an image for the line, where all line pixels are black, rest white
    unsigned char *line_data = new unsigned char[image_width*image_height*3];
    memset(line_data, 255, image_width*image_height*3);
    for (int j = 0; j < line_pixels_count; j++) {
      int pixel_index = lines[i][j];
      line_data[3 * pixel_index + 0] = 0;
      line_data[3 * pixel_index + 1] = 0;
      line_data[3 * pixel_index + 2] = 0;
    }
    char line_no[16];
    sprintf(line_no, "%d", i);
    std::string outfilename = std::string("resources/line") + line_no + "_pixel(100,100).jpg";
    if(!jpge::compress_image_to_jpeg_file(outfilename.c_str(), image_width, image_height, 3, line_data))
    {
      (*message) = "Error while writing image to disk";
      return false;
    }
    delete[] line_data;
  }
  // cleanup
  for (int i = 0; i < line_count_; i++) {
    delete[] lines[i];
    delete[] weights[i];
  }
  delete[] lines;
  delete[] weights;


  // Test Shared adress to x y
  std::vector<int> xs, ys;
  for (int i = 0; i < 42*42; i++) {
    int x, y;
    ImageCoordinatesFromSharedAddress(i, 42, 10, 10, &x, &y);
    xs.push_back(x); ys.push_back(y);
  }
  // print all coordinates
  int y = ys[0];
  char x_string[16], y_string[16];
  *additional_message = "";
  for (int i = 0; i < 42*42; i++) {
    if (ys[i] != y) {
      y = ys[i];
      (*additional_message) += "\n";
    }
    sprintf(x_string, "%d", xs[i]);
    sprintf(y_string, "%d", ys[i]);
    (*additional_message) += std::string("(") + x_string + ", " + y_string + ") ";
  }
  *is_additional_message = false;

  return true;
}
*/
