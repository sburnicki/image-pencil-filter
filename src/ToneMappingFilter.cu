#include "hip/hip_runtime.h"
#include "ToneMappingFilter.h"
#include <cmath>
#include <iostream>

#include "ImagePencilFilter.h"
#define EPSILON 0.0001

// TODO: remove/resolve duplicate code:
__device__ __host__ int PixelIndexOf2(int x, int y, int width) {
	return x + y * width;
}

__device__ __host__ bool IsInImage2(int x, int y, int width, int height) {
	return x >= 0 && x < width &&
			y >= 0 && y < height;
}

// search function
__device__ __host__ int binarySearch(float value, float* target, int minidx, int maxidx) {
    while(true)
    {
    	int pivot = (maxidx - minidx) / 2 + minidx;
    	if (maxidx <= minidx)
    	{
    		return minidx;
    	}
    	float mapval = target[pivot];
    	float diff = std::abs(value - mapval);
    	if (diff < EPSILON)
    	{
    		return pivot;
    	}
    	if (value < mapval)
    	{
    		maxidx = pivot - 1;
    	}
    	else if (value > mapval)
    	{
    		minidx = pivot + 1;
    	}
	}
}

/*
 * Important: This kernel does not only perform tone mapping,
 * but also applies the log2f function to it!
 */
__global__ void ToneMappingKernel(
		float *image,
		float *result,
		int image_width, int image_height,
		int num_tones, int *origHist, float *destHist) {
	// some neat index calculations:
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	float numpixels = image_width * image_height;

	if (IsInImage2(x, y, image_width, image_height)) {
		int pixel_index = PixelIndexOf2(x, y, image_width);
		float find = ((float) origHist[(int) image[pixel_index]]) / numpixels;
		int targetValue = binarySearch(find, destHist, 0, num_tones - 1);
		result[pixel_index] = log2f(targetValue / 255.0);
	}
}


ToneMappingFilter::ToneMappingFilter(ToneMap &destinationMap, int *gpuCumHistogram) : ImageFilter()
{
	gpu_histogram_ = gpuCumHistogram;
	const std::vector<float> &tonemap = destinationMap.getTonemap();
	hipMalloc((void**) &gpu_tonemap_array_, COLOR_DEPTH * sizeof(float));
	hipMemcpy(gpu_tonemap_array_, &tonemap[0], COLOR_DEPTH * sizeof(float), hipMemcpyHostToDevice);
}

ToneMappingFilter::~ToneMappingFilter()
{
	hipFree(gpu_tonemap_array_);
}

void ToneMappingFilter::Run() {
	int imagew = GetImageWidth();
	int imageh = GetImageHeight();
	dim3 thread_block_size(32, 32, 1);
	dim3 block_grid_size(1 + imagew / thread_block_size.x,
			1 + imageh / thread_block_size.y,
			1);
	ToneMappingKernel<<<block_grid_size, thread_block_size>>>(
			GetGpuImageData(),
			GetGpuResultData(),
			imagew, imageh, COLOR_DEPTH,
			gpu_histogram_, gpu_tonemap_array_);
}
