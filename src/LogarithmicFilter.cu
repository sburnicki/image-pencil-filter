#include "hip/hip_runtime.h"
#include "LogarithmicFilter.h"

// TODO: remove/resolve duplicate code:
__device__ __host__ int PixelIndexOf4(int x, int y, int width) {
	return x + y * width;
}

__device__ __host__ bool IsInImage4(int x, int y, int width, int height) {
	return x >= 0 && x < width &&
			y >= 0 && y < height;
}

// TODO: remove the /255.0 operations and decide in which space we are

__global__ void LogarithmicKernel(
		float *img,
		float *result,
		int image_width, int image_height) {
	// some neat index calculations:
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (IsInImage4(x, y, image_width, image_height)) {
		int pixel_index = PixelIndexOf4(x, y, image_width);
		result[pixel_index] = logf(img[pixel_index] / 255.f);
	}
}

LogarithmicFilter::LogarithmicFilter()
{
}

void LogarithmicFilter::Run()
{
	int imagew = GetImageWidth();
	int imageh = GetImageHeight();
	dim3 thread_block_size(32, 32, 1);
	dim3 block_grid_size(1 + imagew / thread_block_size.x,
			1 + imageh / thread_block_size.y,
			1);
	LogarithmicKernel<<<block_grid_size, thread_block_size>>>(
			GetGpuImageData(),
			GetGpuResultData(),
			imagew, imageh);
}
